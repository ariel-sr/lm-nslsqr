// errur.cu
// Implementation file for error.h, containing routines to manage errores
// or return values from the CUDA or CuBLAS functions.

#include <lmnslsqr/error.h>
#include <hipblas.h>
#include <stdio.h>
#include <stdlib.h>

void cublas_check_error(hipblasStatus_t stat, const char *msg)
{
	if (stat != HIPBLAS_STATUS_SUCCESS) {
		printf(msg);
		switch (stat) {
			case HIPBLAS_STATUS_ALLOC_FAILED:
				printf("HIPBLAS_STATUS_ALLOC_FAILED\n");
				break;
			case HIPBLAS_STATUS_INVALID_VALUE:
				printf("HIPBLAS_STATUS_INVALID_VALUE\n");
				break;
			case HIPBLAS_STATUS_ARCH_MISMATCH:
				printf("HIPBLAS_STATUS_ARCH_MISMATCH\n");
				break;
			case HIPBLAS_STATUS_MAPPING_ERROR:
				printf("HIPBLAS_STATUS_MAPPING_ERROR\n");
				break;
			case HIPBLAS_STATUS_EXECUTION_FAILED:
				printf("HIPBLAS_STATUS_EXECUTION_FAILED\n");
				break;
			case HIPBLAS_STATUS_INTERNAL_ERROR:
				printf("HIPBLAS_STATUS_INTERNAL_ERROR\n");
				break;
			case HIPBLAS_STATUS_NOT_SUPPORTED:
				printf("HIPBLAS_STATUS_NOT_SUPPORTED\n");
				break;
			case HIPBLAS_STATUS_UNKNOWN:
				printf("HIPBLAS_STATUS_UNKNOWN\n");
				break;
			default:
				printf("Unknown error\n");
		}
		exit(1);
	}
}

void cuda_check_error(hipError_t err, const char *msg)
{
	if (err != hipSuccess) {
		printf(msg);
		printf("ERROR: %s\n", hipGetErrorString(err));
		exit(0);
	}
}
